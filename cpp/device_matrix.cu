#include "hip/hip_runtime.h"
#include "device_matrix/device_matrix.h"

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
// From http://stackoverflow.com/questions/16077464/atomicadd-for-double-on-gpu.
//
// This is a hack that allows the tests to run in double precision.
// atomicAdd for doubles is available in CUDA 8 and onwards.
__device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

namespace cuda {

hipStream_t merge_streams(const hipStream_t first,
                           const hipStream_t second) {
    if (first == second) {
      return first;
    }

    hipEvent_t first_stream_wait_on_second;
    CCE(hipEventCreate(&first_stream_wait_on_second));
    CCE(hipStreamWaitEvent(first, first_stream_wait_on_second, 0));
    CCE(hipEventRecord(first_stream_wait_on_second, second));
    CCE(hipEventDestroy(first_stream_wait_on_second));

    return first;
}

}  // namespace cuda
