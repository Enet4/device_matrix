#include "device_matrix/device_matrix.h"

namespace cuda {

hipStream_t merge_streams(const hipStream_t first,
                           const hipStream_t second) {
    if (first == second) {
      return first;
    }

    hipEvent_t first_stream_wait_on_second;
    CCE(hipEventCreate(&first_stream_wait_on_second));
    CCE(hipStreamWaitEvent(first, first_stream_wait_on_second, 0));
    CCE(hipEventRecord(first_stream_wait_on_second, second));
    CCE(hipEventDestroy(first_stream_wait_on_second));

    return first;
}

}  // namespace cuda
