#include "device_matrix/runtime.h"

#include <glog/logging.h>

namespace cuda {

template <>
Runtime<FLOATING_POINT_TYPE>* Runtime<FLOATING_POINT_TYPE>::INSTANCE_ = new Runtime;

template <typename FloatT>
Runtime<FloatT>::Runtime() : ZERO(nullptr), ONE(nullptr) {
    int device_count;
    checkCudaErrors(hipGetDeviceCount(&device_count));
    LOG_IF(FATAL, (device_count == 0)) << "Unable to find any CUDA-enabled device.";

    const int32 device_id = 0;

    CHECK_LT(device_id, device_count)
          << "Invalid CUDA device identifier "
          << "(" << device_count << " devices available).";

    CCE(hipSetDevice(device_id));

    // Fresh start.
    CCE(hipDeviceReset());

    // Hard-coded to run on device #0.
    CCE(hipGetDeviceProperties(&props_, device_id));
    CCE(hipblasCreate(&handle_));
    CCE(hipblasSetPointerMode(handle_, HIPBLAS_POINTER_MODE_DEVICE));

    LOG(INFO) << "Using device #" << device_id << ".";

    memset(&device_, 0, sizeof(device_));
    device_.device = device_id;
    device_.size = (size_t) (0.85 * props_.totalGlobalMem);
    CHECK_EQ(CNMEM_STATUS_SUCCESS, cnmemInit(1, &device_, CNMEM_FLAGS_DEFAULT));

    CCE(hipMalloc(const_cast<FloatT**>(&ZERO), sizeof(FloatT)));
    const FloatT zero = 0.0;
    CCE(hipMemcpy(const_cast<FloatT*>(ZERO), &zero,
                   sizeof(FloatT),
                   hipMemcpyHostToDevice));

    CCE(hipMalloc(const_cast<FloatT**>(&ONE), sizeof(FloatT)));
    const FloatT one = 1.0;
    CCE(hipMemcpy(const_cast<FloatT*>(ONE),
                   &one, sizeof(FloatT),
                   hipMemcpyHostToDevice));
}

const decltype(&hipblasSgemm) CuBLAS<float32>::gemm = &hipblasSgemm;
const decltype(&hipblasSgemv) CuBLAS<float32>::gemv = &hipblasSgemv;
const decltype(&hipblasSger) CuBLAS<float32>::ger = &hipblasSger;

const decltype(&hipblasDgemm) CuBLAS<float64>::gemm = &hipblasDgemm;
const decltype(&hipblasDgemv) CuBLAS<float64>::gemv = &hipblasDgemv;
const decltype(&hipblasDger) CuBLAS<float64>::ger = &hipblasDger;

// Explicit instantiation.
template class Runtime<FLOATING_POINT_TYPE>;

}  // namespace cuda
