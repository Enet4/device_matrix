#include <device_matrix/device_matrix.h>

#include <glog/logging.h>
#include <memory>

using namespace cuda;

int main(int argc, char* argv[]) {
    google::InitGoogleLogging(argv[0]);

    const hipStream_t stream = 0; // default CUDA stream.

    std::unique_ptr<device_matrix<float32>> a(
        device_matrix<float32>::create(
            stream,
            {1.0, 2.0, 3.0, 4.0, 5.0, 6.0},
            2 /* num_rows */, 3 /* num_columns */));

    std::unique_ptr<device_matrix<float32>> b(
        device_matrix<float32>::create(
            stream,
            {7.0, 8.0, 9.0, 10.0, 11.0, 12.0},
            3 /* num_rows */, 2 /* num_columns */));

    device_matrix<float32> c(
        2 /* num_rows */, 2 /* num_columns */, stream);

    matrix_mult(stream,
                *a, HIPBLAS_OP_N,
                *b, HIPBLAS_OP_N,
                &c);

    hipDeviceSynchronize();

    print_matrix(c);
}
